#include "hip/hip_runtime.h"
#include <stdio.h>

#include "device_macros.h"

// WrapFFTX.cpp includes AnyFFT.H
#include "WrapFFTX.cpp"

#include "fftx3utilities.h"
#include "sizes.h"

template<typename T>
T minSubarray(const T* arr, int lo, int hi)
{
  T val = arr[lo];
  for (int i = lo+1; i <= hi; i++)
    {
      if (arr[i] < val)
        {
          val = arr[i];
        }
    }
  return val;
}


template<typename T>
T maxSubarray(const T* arr, int lo, int hi)
{
  T val = arr[lo];
  for (int i = lo+1; i <= hi; i++)
    {
      if (arr[i] > val)
        {
          val = arr[i];
        }
    }
  return val;
}


template<typename T>
T avgSubarray(const T* arr, int lo, int hi)
{
  T tot = 0.;
  int len = 0;
  for (int i = lo; i <= hi; i++)
    {
      tot += arr[i];
      len++;
    }
  T avg = tot / (len * 1.);
  return avg;
}

void setRand(double& a_val)
{
  a_val = 1. - ((double) rand()) / (double) (RAND_MAX/2);
}

void setRand(std::complex<double>& a_val)
{
  double x, y;
  setRand(x);
  setRand(y);
  a_val = std::complex<double>(x, y);
}

double diffAbs(double a_x,
               double a_y)
{
  double diffNorm = a_x - a_y;
  if (diffNorm < 0.) diffNorm = -diffNorm;
  return diffNorm;
}

double diffAbs(std::complex<double>& a_x,
               std::complex<double>& a_y)
{
  double diffNorm = std::abs(a_x - a_y);
  return diffNorm;
}

DEVICE_FFT_RESULT deviceExecD2Z(DEVICE_FFT_HANDLE a_plan,
                                double* a_in,
                                std::complex<double>* a_out)
{
  return DEVICE_FFT_EXECD2Z(a_plan,
                            (DEVICE_FFT_DOUBLEREAL*) a_in,
                            (DEVICE_FFT_DOUBLECOMPLEX*) a_out);
}


template<typename T_IN, typename T_OUT>
struct deviceTransform
{
  deviceTransform(DEVICE_FFT_TYPE a_tp,
                  int a_dir = 0)
  {
    m_tp = a_tp;
    m_dir = a_dir;
  }
                  
  DEVICE_FFT_TYPE m_tp;

  int m_dir;

  DEVICE_FFT_RESULT exec(DEVICE_FFT_HANDLE a_plan,
                         T_IN* a_in,
                         T_OUT* a_out)
  {
    if (m_tp == DEVICE_FFT_Z2Z)
      {
        return DEVICE_FFT_EXECZ2Z(a_plan,
                                  (DEVICE_FFT_DOUBLECOMPLEX*) a_in,
                                  (DEVICE_FFT_DOUBLECOMPLEX*) a_out,
                                  m_dir);
      }
    else if (m_tp == DEVICE_FFT_D2Z)
      {
        return DEVICE_FFT_EXECD2Z(a_plan,
                                  (DEVICE_FFT_DOUBLEREAL*) a_in,
                                  (DEVICE_FFT_DOUBLECOMPLEX*) a_out);
      }
    else if (m_tp == DEVICE_FFT_Z2D)
      {
        return DEVICE_FFT_EXECZ2D(a_plan,
                                  (DEVICE_FFT_DOUBLECOMPLEX*) a_in,
                                  (DEVICE_FFT_DOUBLEREAL*) a_out);
      }
    else
      {
        return (DEVICE_FFT_RESULT) -1;
      }
  }
};
  

deviceTransform<double, std::complex<double> >
mdprdftDevice(DEVICE_FFT_D2Z);

deviceTransform<std::complex<double>, double>
imdprdftDevice(DEVICE_FFT_Z2D);


template<typename T_IN, typename T_OUT>
void compareSize(const fftx::point_t<3>& a_sz,
                 AnyFFT::direction a_dir,
                 deviceTransform<T_IN, T_OUT>& a_tfmDevice,
                 int a_verbosity)
{
  /*
    Allocate space for arrays, and set input array.
  */
  // const fftx::point_t<3> realSize = a_sz;
  // const fftx::point_t<3> complexSize = fftx::point_t<3>({{a_sz[0]/2 + 1, a_sz[1], a_sz[2]}});

  // This doesn't work. :/
  // const fftx::point_t<3> unit = fftx::point_t<3>::Unit();
  //   fftx::box_t<3> inputDomain(unit, inputSize);
  //   fftx::box_t<3> outputDomain(unit, outputSize);

  // N.B.: in transform, real size is {tfm_sz[0], tfm_sz[1], tfm_sz[2]}
  // and complex size is {tfm_sz[0]/2+1, tfm_sz[1], tfm_sz[2]},
  // where tfm_sz is the size of the real domain within the FFTX transform,
  // which has the reverse of the dimensions in CreatePlan.
  // Because of the reversals, here we define the domains with reversals.
  // The only time we use realDomain or complexDomain here other than to get
  // the number of points in them is when we look up the indices of where
  // there are differences.
  fftx::box_t<3> realDomain(fftx::point_t<3>({{1, 1, 1}}),
                            fftx::point_t<3>({{a_sz[2], a_sz[1], a_sz[0]}}));
  // The only reason 
  fftx::box_t<3> complexDomain(fftx::point_t<3>({{1, 1, 1}}),
                               fftx::point_t<3>({{a_sz[2]/2+1, a_sz[1], a_sz[0]}}));
  
  fftx::array_t<3, double> realArrayHost(realDomain);
  fftx::array_t<3, std::complex<double>> complexArrayHost(complexDomain);
  if (a_dir == AnyFFT::direction::R2C)
    {
      forall([](double(&v), const fftx::point_t<3>& p)
             {
               setRand(v);
             }, realArrayHost);
    }
  else if (a_dir == AnyFFT::direction::C2R)
    {
      forall([](std::complex<double>(&v), const fftx::point_t<3>& p)
             {
               setRand(v);
             }, complexArrayHost);
    }
  else
    {
      std::cout << "direction must be either R2C or C2R" << std::endl;
      exit(-1);
    }
    
  /*
    Define the FFTX transform.
  */
  std::cout << "Creating plan "
            << ((a_dir == AnyFFT::direction::R2C) ? "R2C" : "C2R")
            << " on size " << a_sz << std::endl;
  AnyFFT::FFTplan this_fftx_plan =
    AnyFFT::CreatePlan(// amrex::IntVect(AMREX_D_DECL(a_sz[0], a_sz[1], a_sz[2])),
                       a_sz,
                       realArrayHost.m_data.local(),
                       complexArrayHost.m_data.local(),
                       a_dir,
                       3);

  /*
    Execute the FFTX transform.
  */
  if (this_fftx_plan.m_plan.defined())
    {
      std::cout << "Executing plan" << std::endl;
      AnyFFT::Execute(this_fftx_plan);

      /*
        Define the device transform.
      */
      DEVICE_FFT_HANDLE this_device_plan;
      {
        std::cout << "Defining plan for device transform" << std::endl;
        auto result =
          DEVICE_FFT_PLAN3D(&this_device_plan, a_sz[2], a_sz[1], a_sz[0],
                            a_tfmDevice.m_tp);
        if (result != DEVICE_FFT_SUCCESS)
          {
            std::cout << "deviceFFT plan define failed\n" << std::endl;
            exit(-1);
          }
      }

      /*
        Execute the device transform, and compare result with that from FFTX.
      */
      fftx::box_t<3> inputDomain, outputDomain;
      T_IN* inputHostPtr;
      T_OUT* outputFFTXHostPtr;
      std::string tfmName;
      if (a_dir == AnyFFT::direction::R2C)
        {
          inputDomain = realDomain;
          outputDomain = complexDomain;
          inputHostPtr = (T_IN*) realArrayHost.m_data.local();
          outputFFTXHostPtr = (T_OUT*) complexArrayHost.m_data.local();
          tfmName = this_fftx_plan.m_plan.m_tfm_3d_r2c->name();
        }
      else if (a_dir == AnyFFT::direction::C2R)
        {
          inputDomain = complexDomain;
          outputDomain = realDomain;
          inputHostPtr = (T_IN*) complexArrayHost.m_data.local();
          outputFFTXHostPtr = (T_OUT*) realArrayHost.m_data.local();
          tfmName = this_fftx_plan.m_plan.m_tfm_3d_c2r->name();
        }
      else
        {
          std::cout << "direction must be either R2C or C2R" << std::endl;
          exit(-1);
        }
      
      {
        auto input_bytes = inputDomain.size() * sizeof(T_IN);
        auto output_bytes = outputDomain.size() * sizeof(T_OUT);
        
        char* bufferDevicePtr;
        DEVICE_MALLOC(&bufferDevicePtr, input_bytes + output_bytes);
        T_IN* inputDevicePtr = (T_IN*) bufferDevicePtr;
        bufferDevicePtr += input_bytes;
        T_OUT* outputDevicePtr = (T_OUT*) bufferDevicePtr;
        
        // Have already set realArrayHost.
        DEVICE_MEM_COPY(inputDevicePtr, // dest
                        inputHostPtr, // source
                        input_bytes, // bytes
                        MEM_COPY_HOST_TO_DEVICE); // type
        
        auto result = a_tfmDevice.exec(this_device_plan,
                                       inputDevicePtr,
                                       outputDevicePtr);
        if (result != DEVICE_FFT_SUCCESS)
          {
            std::cout << "deviceFFTExec launch failed\n" << std::endl;
            exit(-1);
          }
        
        auto nptsOutput = outputDomain.size();
        T_OUT* outputHostPtr = new T_OUT[nptsOutput];
        DEVICE_MEM_COPY(outputHostPtr, // dest
                        outputDevicePtr, // source
                        output_bytes, // bytes
                        MEM_COPY_DEVICE_TO_HOST); // type
        DEVICE_FREE(bufferDevicePtr);
        
        // Now find differences between outputHostPtr
        // and complexArrayHost.m_data.local().
        const double tol = 1.e-7;
        bool match = true;
        double maxDiff = 0.;
        for (size_t ind = 0; ind < nptsOutput; ind++)
          {
            T_OUT outputFFTXPoint = outputFFTXHostPtr[ind];
            T_OUT outputDeviceFFTPoint = outputHostPtr[ind];
            double diffAbsPoint = diffAbs(outputFFTXPoint, outputDeviceFFTPoint);
            updateMaxAbs(maxDiff, diffAbsPoint);
            bool matchPoint = (diffAbsPoint < tol);
            if (!matchPoint)
              {
                match = false;
                if (a_verbosity >= 3)
                  {
                    fftx::point_t<3> pt = pointFromPositionBox(ind, outputDomain);
                    // Take the flipped indices because remember we
                    // flipped the dimensions of the domains.
                    std::cout << "error at " << pt.flipped()
                              << ": FFTX " << outputFFTXPoint
                              << ", deviceFFT " << outputDeviceFFTPoint
                              << std::endl;
                  }
              }
          }
        delete[] outputHostPtr;
        if (match)
          {
            printf("YES, results match for %s. Max diff %11.5e\n",
                   tfmName.c_str(), maxDiff);
          }
        else
          {
            printf("NO, results do not match for %s. Max diff %11.5e\n",
                   tfmName.c_str(), maxDiff);
          }
      }
    }
  
  /*
    Destroy the FFTX transform.
  */
  std::cout << "Destroying plan" << std::endl;
  AnyFFT::DestroyPlan(this_fftx_plan);
}


int main(int argc, char* argv[])
{
  printf("Usage:  %s [verbosity=0]\n", argv[0]);
  // printf("verbosity 0 for avg times, 1 for min/max, 2 for all iterations, 3 for errors\n");
  int verbosity = 0;
  if (argc > 1)
    {
      verbosity = atoi(argv[1]);
    }
  printf("Running with verbosity %d\n", verbosity);

  // last entry is { 0, 0, 0 }
  int numentries = sizeof ( AllSizes3 ) / sizeof ( fftx::point_t<3> ) - 1;

  for ( int ind = 0; ind < numentries; ind++ )
    {
      fftx::point_t<3> sz = AllSizes3[ind];

      compareSize(sz, AnyFFT::direction::R2C, mdprdftDevice, verbosity);

      compareSize(sz, AnyFFT::direction::C2R, imdprdftDevice, verbosity);
    }
  
  printf("%s: All done, exiting\n", argv[0]);
  return 0;
}
